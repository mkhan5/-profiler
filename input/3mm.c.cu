
/* Header below added by Tulsi for replaced CUBLAS code */
#include <hip/hip_runtime.h>
#include <hipblas.h>


#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
//#define N 3
int N=3;

void mm( double A[N][N],
            double B[N][N],
            double C[N][N],
            double D[N][N],
            double E[N][N],
            double tmp1[N][N],
            double tmp2[N][N]
           )
{
  int i, j,k;

   /* tmp1 := A*B */
   
    hipblasStatus_t status;
    hipblasHandle_t handle;
    double *d_A = 0;
    double *d_B = 0;
    double *d_C = 0;
    double *d_D = 0;
    double *d_E = 0;
    double *d_tmpAB = 0;
    double *d_tmpCD = 0;
    const double cublas_alpha = 1.0f;
    const double cublas_beta = 0.0f;
    hipblasCreate(&handle);
    
    hipMalloc((void **)&d_A, N * N * sizeof(d_A[0]));
    hipMalloc((void **)&d_B, N * N * sizeof(d_B[0]));
    hipMalloc((void **)&d_C, N * N * sizeof(d_C[0]));
    hipMalloc((void **)&d_D, N * N * sizeof(d_D[0]));
    hipMalloc((void **)&d_E, N * N * sizeof(d_E[0]));
    hipMalloc((void **)&d_tmpAB, N * N * sizeof(d_tmpAB[0]));
    hipMalloc((void **)&d_tmpCD, N * N * sizeof(d_tmpCD[0]));
    
    hipMemcpy(d_A, A,  N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N *N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, N *N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_D, D, N *N * sizeof(double), hipMemcpyHostToDevice);
    
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &cublas_alpha, d_B, N, d_A, N, &cublas_beta, d_tmpAB, N);
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &cublas_alpha, d_D, N, d_C, N, &cublas_beta, d_tmpCD, N);
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &cublas_alpha, d_tmpCD, N, d_tmpAB, N, &cublas_beta, d_E, N);
    hipMemcpy(E, d_E, N*N*sizeof(double), hipMemcpyDeviceToHost);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_D);
    hipFree(d_E);
    hipFree(d_tmpAB);
    hipFree(d_tmpCD);
    hipblasDestroy(handle);


}



int main(int argc, char** argv)
{

  int M = 3;


    double a[3][3] = { 1, 2, 5, 9, 23, -7, 6, 1, -3};
    double b[3][3] = {1, 5, -1, -2, 1, 21, -30, -11, -15};
    double c[3][3] = {1, 5, -1, -2, 1, 21, -30, -11, -15};
    double d[3][3] = { 1, 2, 5, 9, 23, -7, 6, 1, -3};

//    double a[3][3] = { 24, -22, -4, -11, 12, 10, -23, -18, -12};
//    double b[3][3] = { 20, -25, -1, -2, -3, 14, -24, -9, -19};
//    double c[3][3] = {-17, -16, 19, 8, 5, 4, 17, -13, 22};
//    double d[3][3] = {-5, -8, -6, 1, -15, 23, 3, -21, 11};

  double tmp1[3][3];
  double tmp2[3][3];
  double e[3][3];
  //sol: y = (13, 31, 49)
  double alpha = 1;
  double beta = 0;

  mm(a, b, c, d, e, tmp1, tmp2);
  //std::copy( res, res + 3, std::ostream_iterator<float>( std::cout, ","));
  int i,j;
  printf("The res E = tmp1.tmp2 is \n");
  for( i = 0; i < N; i++)
    {
        for (j = 0; j < N; j++)
      {
        printf(" %f ",e[i][j]);
       }
       printf("\n");
    }

  printf("\n");

}
