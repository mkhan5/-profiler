#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Header below added by Tulsi */
#include <stdlib.h>


/* Header below added by Qahwa for replaced CUBLAS code */
#include <hipblas.h>

int N = 3;
void symm(float alpha, float beta, float A[N][N], float B[N][N], float C[N][N])
{
  int i;
  int j;
  int k;
  float temp2;
  float *a_d;
  float *x_d;
  hipMalloc(&a_d, (N * N) * (sizeof(float)));
  hipMalloc(&b_d, (N * N) * (sizeof(float)));
  hipMemcpy(a_d, A, (N * N) * (sizeof(float)), hipMemcpyHostToDevice);
  hipMemcpy(b_d, B, (N * N) * (sizeof(float)), hipMemcpyHostToDevice);
  const float alpha = 1.0f;
  const float beta = 0.0f;
  hipblasSsymm(CblasRowMajor, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER, N, N, alpha, A, N, B, N, beta, C, N);
  hipMemcpy(c, C, (N * N) * (sizeof(float)), hipMemcpyDeviceToHost);
  hipFree(a_d);
  hipFree(b_d);
  hipblasDestroy(handle);
}

int main(int argc, char **argv)
{
  int M = 3;
  int n = N;
  float alpha = 1;
  float beta = 0;
  float a[3][3] = {1, 0, 0, 2, 5, 0, 3, 6, -3};
  float b[3][3] = {1, 0, 1, 3, -1, -2, 0, 2, -1};
  float res[3][3];
  symm(alpha, beta, a, b, res);
  int i;
  int j;
  printf("The res x is \n");
  for (i = 0; i < N; i++)
  {
    for (j = 0; j < N; j++)
    {
      printf(" %f ", res[i][j]);
    }

    printf("\n");
  }

  printf("\n");
}

