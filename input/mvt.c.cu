#include "hip/hip_runtime.h"

/* Header below added by Tulsi for replaced CUBLAS code */
#include <hip/hip_runtime.h>
#include <hipblas.h>


#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>


//#define N 3
int N=3;

void mvt( double A[N][N],
           double x1[N],
           double x2[N],
           double y_1[N],
           double y_2[N])
{
  int i, j;



    hipblasStatus_t status;
    hipblasHandle_t handle;
    double *d_A = 0;
    double *x1 = 0;
    double *x2 = 0;
    double *y1 = 0;
    double *y2 = 0;
    const double cublas_alpha = 1.0;
    const double cublas_beta = 1.0;

    hipblasCreate(&handle);
    hipMalloc((void **)&d_A, N * N * sizeof(d_A[0]));
    hipMalloc((void **)&x1,  N * sizeof(x1[0]));
    hipMalloc((void **)&x2,  N * sizeof(x2[0]));
    hipMalloc((void **)&y1,  N * sizeof(y1[0]));
    hipMalloc((void **)&y2,  N * sizeof(y2[0]));

    hipMemcpy(d_A, A,  N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(x1, x1, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(x2, x2, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(y1, y_1, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(y2, y_2, N * sizeof(double), hipMemcpyHostToDevice);


    hipblasDgemv(handle, HIPBLAS_OP_T, N, N, &cublas_alpha, d_A, N, y1, 1, &cublas_beta, x1, 1);
    hipblasDgemv(handle, HIPBLAS_OP_N, N, N, &cublas_alpha, d_A, N, y2, 1, &cublas_beta, x2, 1);
    hipMemcpy( x1, x1, N*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy( x2, x2, N*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(x1);
    hipFree(x2);
    hipFree(y1);
    hipFree(y2);
    hipblasDestroy(handle);

}


int main(int argc, char** argv)
{

  int M = 4;
  int i,j;

  //double a[4][4] = { 1, 2, 3, 4, 5, 6, 7, 8, 9, 6, 7, 8};
  //double x[4] = { 2, 1, 3, 5};
  double a[3][3] = { 1, -2, 3, 4, 6, 6, -17, -11, 8};
  double x1[3] = { -5, -9, 6};
  double x2[3] = { -2, 3, 21};
  double y1[3] = { -1, -4, 23};
  double y2[3] = { 7, -8, -19};

printf("The x1 is \n");
  for( i = 0; i < N; i++)
    {
      printf(" %f ",x1[i]);
    }

  printf("\n");
  printf("The x2 is \n");
  for( i = 0; i < N; i++)
    {
      printf(" %f ",x2[i]);
    }

  printf("\n");

    mvt(a, x1, x2, y1, y2);
  //std::copy( res, res + 3, std::ostream_iterator<double>( std::cout, ","));

  printf("The res x1 is \n");
  for( i = 0; i < N; i++)
    {
      printf(" %f ",x1[i]);
    }

  printf("\n");
  printf("The res x2 is \n");
  for( i = 0; i < N; i++)
    {
      printf(" %f ",x2[i]);
    }

  printf("\n");

}
