#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <polybench.h>
#include "trisolv.h"

/* Header below added by Tulsi */
#include <stdlib.h>


/* Header below added by Qahwa for replaced CUBLAS code */
#include <hipblas.h>

extern void *polybench_alloc_data(unsigned long long int n, int elt_size);
static void init_array(int n, double A[4000 + 0][4000 + 0], double x[4000 + 0], double c[4000 + 0])
{
  int i;
  int j;
  for (i = 0; i < n; i++)
  {
    c[i] = (x[i] = ((double) i) / n);
    for (j = 0; j < n; j++)
      A[i][j] = (((double) i) * j) / n;

  }

}

static void print_array(int n, double x[4000 + 0])
{
  int i;
  for (i = 0; i < n; i++)
  {
    fprintf(stderr, "%0.2lf ", x[i]);
    if ((i % 20) == 0)
      fprintf(stderr, "\n");

  }

}

static void kernel_trisolv(int n, double A[4000 + 0][4000 + 0], double x[4000 + 0], double c[4000 + 0])
{
  int i;
  int j;
  float *a_d;
  float *b_d;
  hipMalloc(&a_d, (n * n) * (sizeof(float)));
  hipMalloc(&b_d, n * (sizeof(float)));
  hipMemcpy(b_d, c, n * (sizeof(float)), hipMemcpyHostToDevice);
  hipMemcpy(a_d, A, (n * n) * (sizeof(float)), hipMemcpyHostToDevice);
  const float alpha = 1.0f;
  hipblasStrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, 1, &alpha, A, n, c, n);
  hipMemcpy(x, b_d, n * (sizeof(float)), hipMemcpyDeviceToHost);
  hipFree(A_d);
  hipFree(b_d);
  hipblasDestroy(handle);
}

int main(int argc, char **argv)
{
  int n = 4000;
  double (*A)[4000 + 0][4000 + 0];
  A = (double (*)[4000 + 0][4000 + 0]) polybench_alloc_data((4000 + 0) * (4000 + 0), sizeof(double));
  ;
  double (*x)[4000 + 0];
  x = (double (*)[4000 + 0]) polybench_alloc_data(4000 + 0, sizeof(double));
  ;
  double (*c)[4000 + 0];
  c = (double (*)[4000 + 0]) polybench_alloc_data(4000 + 0, sizeof(double));
  ;
  init_array(n, *A, *x, *c);
  ;
  kernel_trisolv(n, *A, *x, *c);
  ;
  ;
  if ((argc > 42) && (!strcmp(argv[0], "")))
    print_array(n, *x);

  free((void *) A);
  ;
  free((void *) x);
  ;
  free((void *) c);
  ;
  return 0;
}

