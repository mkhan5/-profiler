#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <polybench.h>
#include "atax.h"

/* Header below added by Tulsi */
#include <stdlib.h>


/* Header below added by Qahwa for replaced CUBLAS code */
#include <hipblas.h>

extern void *polybench_alloc_data(unsigned long long int n, int elt_size);
static void init_array(int nx, int ny, double A[4000 + 0][4000 + 0], double x[4000 + 0])
{
  int i;
  int j;
  for (i = 0; i < ny; i++)
    x[i] = i * M_PI;

  for (i = 0; i < nx; i++)
    for (j = 0; j < ny; j++)
    A[i][j] = (((double) i) * (j + 1)) / nx;


}

static void print_array(int nx, double y[4000 + 0])
{
  int i;
  for (i = 0; i < nx; i++)
  {
    fprintf(stderr, "%0.2lf ", y[i]);
    if ((i % 20) == 0)
      fprintf(stderr, "\n");

  }

  fprintf(stderr, "\n");
}

static void kernel_atax(int nx, int ny, double A[4000 + 0][4000 + 0], double x[4000 + 0], double y[4000 + 0], double tmp[4000 + 0])
{
  int i;
  int j;
  for (i = 0; i < nx; i++)
    y[i] = 0;

  float *a_d;
  float *x_d;
  hipMalloc(&a_d, (nx * nx) * (sizeof(float)));
  hipMalloc(&x_d, nx * (sizeof(float)));
  hipMemcpy(x_d, x, nx * (sizeof(float)), hipMemcpyHostToDevice);
  hipMemcpy(a_d, A, (nx * nx) * (sizeof(float)), hipMemcpyHostToDevice);
  const float alpha = 1.0f;
  const float beta = 0.0;
  hipblasSgemv(handle, HIPBLAS_OP_T, nx, nx, &alpha, A, nx, x, 1, &beta, y, 1);
  hipblasSgemv(handle, HIPBLAS_OP_N, nx, nx, &alpha, A, nx, y, 1, &beta, _result[1], 1);
  hipMemcpy(y, y, nx * (sizeof(float)), hipMemcpyDeviceToHost);
  hipFree(a_d);
  hipFree(x_d);
  hipblasDestroy(handle);
}

int main(int argc, char **argv)
{
  int nx = 4000;
  int ny = 4000;
  double (*A)[4000 + 0][4000 + 0];
  A = (double (*)[4000 + 0][4000 + 0]) polybench_alloc_data((4000 + 0) * (4000 + 0), sizeof(double));
  ;
  double (*x)[4000 + 0];
  x = (double (*)[4000 + 0]) polybench_alloc_data(4000 + 0, sizeof(double));
  ;
  double (*y)[4000 + 0];
  y = (double (*)[4000 + 0]) polybench_alloc_data(4000 + 0, sizeof(double));
  ;
  double (*tmp)[4000 + 0];
  tmp = (double (*)[4000 + 0]) polybench_alloc_data(4000 + 0, sizeof(double));
  ;
  init_array(nx, ny, *A, *x);
  ;
  kernel_atax(nx, ny, *A, *x, *y, *tmp);
  ;
  ;
  if ((argc > 42) && (!strcmp(argv[0], "")))
    print_array(nx, *y);

  free((void *) A);
  ;
  free((void *) x);
  ;
  free((void *) y);
  ;
  free((void *) tmp);
  ;
  return 0;
}

