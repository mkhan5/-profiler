
/* Header below added by Tulsi for replaced CUBLAS code */
#include <hip/hip_runtime.h>
#include <hipblas.h>


/**
 * This version is stamped on May 10, 2016
 *
 * Contact:
 *   Louis-Noel Pouchet <pouchet.ohio-state.edu>
 *   Tomofumi Yuki <tomofumi.yuki.fr>
 *
 * Web address: http://polybench.sourceforge.net
 */
/* bicg.c: this file is part of PolyBench/C */

#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
#include <polybench.h>

/* Include benchmark-specific header. */
#include "bicg.h"


/* Array initialization. */
static
void init_array (int m, int n,
		 DATA_TYPE POLYBENCH_2D(A,N,M,n,m),
		 DATA_TYPE POLYBENCH_1D(r,N,n),
		 DATA_TYPE POLYBENCH_1D(p,M,m))
{
  int i, j;

  for (i = 0; i < m; i++)
    p[i] = (DATA_TYPE)(i % m) / m;
  for (i = 0; i < n; i++) {
    r[i] = (DATA_TYPE)(i % n) / n;
    for (j = 0; j < m; j++)
      A[i][j] = (DATA_TYPE) (i*(j+1) % n)/n;
  }
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int m, int n,
		 DATA_TYPE POLYBENCH_1D(s,M,m),
		 DATA_TYPE POLYBENCH_1D(q,N,n))

{
  int i;

  POLYBENCH_DUMP_START;
  POLYBENCH_DUMP_BEGIN("s");
  for (i = 0; i < m; i++) {
    if (i % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
    fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, s[i]);
  }
  POLYBENCH_DUMP_END("s");
  POLYBENCH_DUMP_BEGIN("q");
  for (i = 0; i < n; i++) {
    if (i % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
    fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, q[i]);
  }
  POLYBENCH_DUMP_END("q");
  POLYBENCH_DUMP_FINISH;
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_bicg(int m, int n,
		 DATA_TYPE POLYBENCH_2D(A,N,M,n,m),
		 DATA_TYPE POLYBENCH_1D(s,M,m),
		 DATA_TYPE POLYBENCH_1D(q,N,n),
		 DATA_TYPE POLYBENCH_1D(p,M,m),
		 DATA_TYPE POLYBENCH_1D(r,N,n))
{
  int i, j;

//#pragma scop
  for (i = 0; i < n; i++)
    s[i] = 0;

    hipblasStatus_t status;
    hipblasHandle_t handle;
    double *d_A = 0;
    double *d_s = 0;
    double *d_q = 0;
    double *d_p = 0;
    double *d_r = 0;
    const double cublas_alpha = 1.0;
    const double cublas_beta = 0.0;

    hipblasCreate(&handle);
    hipMalloc((void **)&d_A, n * n * sizeof(d_A[0]));
    hipMalloc((void **)&d_s,  n * sizeof(d_s[0]));
    hipMalloc((void **)&d_q,  n * sizeof(d_q[0]));
    hipMalloc((void **)&d_p,  n * sizeof(d_p[0]));
    hipMalloc((void **)&d_r,  n * sizeof(d_r[0]));

    hipMemcpy(d_A, A,  n * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_p, p, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_r, r, n * sizeof(double), hipMemcpyHostToDevice);


    hipblasDgemv(handle, HIPBLAS_OP_T, n, n, &cublas_alpha, d_A, n, d_p, 1, &cublas_beta, d_q, 1);
    hipblasDgemv(handle, HIPBLAS_OP_N, n, n, &cublas_alpha, d_A, n, d_r, 1, &cublas_beta, d_s, 1);
    hipMemcpy( q, d_q, n*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy( s, d_s, n*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_s);
    hipFree(d_q);
    hipFree(d_p);
    hipFree(d_r);
    hipblasDestroy(handle);

}


int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int n = N;
  int m = M;

  /* Variable declaration/allocation. */
  POLYBENCH_2D_ARRAY_DECL(A, DATA_TYPE, N, M, n, m);
  POLYBENCH_1D_ARRAY_DECL(s, DATA_TYPE, M, m);
  POLYBENCH_1D_ARRAY_DECL(q, DATA_TYPE, N, n);
  POLYBENCH_1D_ARRAY_DECL(p, DATA_TYPE, M, m);
  POLYBENCH_1D_ARRAY_DECL(r, DATA_TYPE, N, n);

  /* Initialize array(s). */
  init_array (m, n,
	      POLYBENCH_ARRAY(A),
	      POLYBENCH_ARRAY(r),
	      POLYBENCH_ARRAY(p));

  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  kernel_bicg (m, n,
	       POLYBENCH_ARRAY(A),
	       POLYBENCH_ARRAY(s),
	       POLYBENCH_ARRAY(q),
	       POLYBENCH_ARRAY(p),
	       POLYBENCH_ARRAY(r));

  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  polybench_prevent_dce(print_array(m, n, POLYBENCH_ARRAY(s), POLYBENCH_ARRAY(q)));

  /* Be clean. */
  POLYBENCH_FREE_ARRAY(A);
  POLYBENCH_FREE_ARRAY(s);
  POLYBENCH_FREE_ARRAY(q);
  POLYBENCH_FREE_ARRAY(p);
  POLYBENCH_FREE_ARRAY(r);

  return 0;
}
