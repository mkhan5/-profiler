
/* Header below added by Tulsi for replaced CUBLAS code */
#include "hip/hip_runtime.h"
#include <hipsolver.h>
#include <hipblas.h>


/**
 * This version is stamped on May 10, 2016
 *
 * Contact:
 *   Louis-Noel Pouchet <pouchet.ohio-state.edu>
 *   Tomofumi Yuki <tomofumi.yuki.fr>
 *
 * Web address: http://polybench.sourceforge.net
 */
/* lu.c: this file is part of PolyBench/C */

#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
#include <polybench.h>

/* Include benchmark-specific header. */
#include "lu.h"


/* Array initialization. */
static
void init_array (int n,
		 DATA_TYPE POLYBENCH_2D(A,N,N,n,n))
{
  int i, j;

  for (i = 0; i < n; i++)
    {
      for (j = 0; j <= i; j++)
	A[i][j] = (DATA_TYPE)(-j % n) / n + 1;
      for (j = i+1; j < n; j++) {
	A[i][j] = 0;
      }
      A[i][i] = 1;
    }

  /* Make the matrix positive semi-definite. */
  /* not necessary for LU, but using same code as cholesky */
  int r,s,t;
  POLYBENCH_2D_ARRAY_DECL(B, DATA_TYPE, N, N, n, n);
  for (r = 0; r < n; ++r)
    for (s = 0; s < n; ++s)
      (POLYBENCH_ARRAY(B))[r][s] = 0;
  for (t = 0; t < n; ++t)
    for (r = 0; r < n; ++r)
      for (s = 0; s < n; ++s)
	(POLYBENCH_ARRAY(B))[r][s] += A[r][t] * A[s][t];
    for (r = 0; r < n; ++r)
      for (s = 0; s < n; ++s)
	A[r][s] = (POLYBENCH_ARRAY(B))[r][s];
  POLYBENCH_FREE_ARRAY(B);

}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int n,
		 DATA_TYPE POLYBENCH_2D(A,N,N,n,n))

{
  int i, j;

  POLYBENCH_DUMP_START;
  POLYBENCH_DUMP_BEGIN("A");
  for (i = 0; i < n; i++)
    for (j = 0; j < n; j++) {
      if ((i * n + j) % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
      fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, A[i][j]);
    }
  POLYBENCH_DUMP_END("A");
  POLYBENCH_DUMP_FINISH;
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_lu(int n,
	       DATA_TYPE POLYBENCH_2D(A,N,N,n,n))
{
  int i, j, k;

//#pragma scop
    double *d_A,*d_B,*d_AT;
    const double cublas_alpha = 1.0;
    const double cublas_beta = 0.0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipMalloc((void **)&d_A, n * n * sizeof(double));
    hipMalloc((void **)&d_AT, n * n * sizeof(double));

    hipMemcpy(d_A, A, n * n * sizeof(double), hipMemcpyHostToDevice);
    hipblasDgeam(handle,HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, &cublas_alpha, d_A, n, &cublas_beta, d_B, n, d_AT, n);

    int worksize = 0;
    int *devInfo;
    hipMalloc((void **)&devInfo, sizeof(int));
    hipsolverHandle_t solver_handle;
    hipsolverDnCreate(&solver_handle);

    hipsolverDnDgetrf_bufferSize(solver_handle, n, n, d_AT, n, &worksize);
    double *work;
    hipMalloc((void **)&work, worksize * sizeof(double));
    int *devIpiv;
    hipMalloc((void **)&devIpiv, n * sizeof(int));

    hipsolverDnDgetrf(solver_handle, n, n, d_AT, n, work, devIpiv, devInfo);
    int devInfo_h = 0;
    hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    if (devInfo_h != 0)
        printf("Unsuccessful getrf execution\n\n");
    printf("\nFactorized matrix\n");

    hipblasDgeam(handle,HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, &cublas_alpha, d_AT, n, &cublas_beta, d_B, n, d_A, n);
    hipMemcpy(A, d_A, n * n * sizeof(double), hipMemcpyDeviceToHost);
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_AT);
    hipFree(devInfo);
    hipFree(work);
    hipFree(devIpiv);
    hipsolverDnDestroy(solver_handle);
//#pragma endscop
}


int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int n = N;

  /* Variable declaration/allocation. */
  POLYBENCH_2D_ARRAY_DECL(A, DATA_TYPE, N, N, n, n);

  /* Initialize array(s). */
  init_array (n, POLYBENCH_ARRAY(A));

  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  kernel_lu (n, POLYBENCH_ARRAY(A));

  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  polybench_prevent_dce(print_array(n, POLYBENCH_ARRAY(A)));

  /* Be clean. */
  POLYBENCH_FREE_ARRAY(A);

  return 0;
}
