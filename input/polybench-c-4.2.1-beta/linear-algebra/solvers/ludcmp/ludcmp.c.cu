
/* Header below added by Tulsi for replaced CUBLAS code */
#include <hip/hip_runtime.h>
#include <hipblas.h>
/**
 * This version is stamped on May 10, 2016
 *
 * Contact:
 *   Louis-Noel Pouchet <pouchet.ohio-state.edu>
 *   Tomofumi Yuki <tomofumi.yuki.fr>
 *
 * Web address: http://polybench.sourceforge.net
 */
/* ludcmp.c: this file is part of PolyBench/C */

#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
#include <polybench.h>

/* Include benchmark-specific header. */
#include "ludcmp.h"


/* Array initialization. */
static
void init_array (int n,
		 DATA_TYPE POLYBENCH_2D(A,N,N,n,n),
		 DATA_TYPE POLYBENCH_1D(b,N,n),
		 DATA_TYPE POLYBENCH_1D(x,N,n),
		 DATA_TYPE POLYBENCH_1D(y,N,n))
{
  int i, j;
  DATA_TYPE fn = (DATA_TYPE)n;

  for (i = 0; i < n; i++)
    {
      x[i] = 0;
      y[i] = 0;
      b[i] = (i+1)/fn/2.0 + 4;
    }

  for (i = 0; i < n; i++)
    {
      for (j = 0; j <= i; j++)
	A[i][j] = (DATA_TYPE)(-j % n) / n + 1;
      for (j = i+1; j < n; j++) {
	A[i][j] = 0;
      }
      A[i][i] = 1;
    }

  /* Make the matrix positive semi-definite. */
  /* not necessary for LU, but using same code as cholesky */
  int r,s,t;
  POLYBENCH_2D_ARRAY_DECL(B, DATA_TYPE, N, N, n, n);
  for (r = 0; r < n; ++r)
    for (s = 0; s < n; ++s)
      (POLYBENCH_ARRAY(B))[r][s] = 0;
  for (t = 0; t < n; ++t)
    for (r = 0; r < n; ++r)
      for (s = 0; s < n; ++s)
	(POLYBENCH_ARRAY(B))[r][s] += A[r][t] * A[s][t];
    for (r = 0; r < n; ++r)
      for (s = 0; s < n; ++s)
	A[r][s] = (POLYBENCH_ARRAY(B))[r][s];
  POLYBENCH_FREE_ARRAY(B);

}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int n,
		 DATA_TYPE POLYBENCH_1D(x,N,n))

{
  int i;

  POLYBENCH_DUMP_START;
  POLYBENCH_DUMP_BEGIN("x");
  for (i = 0; i < n; i++) {
    if (i % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
    fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, x[i]);
  }
  POLYBENCH_DUMP_END("x");
  POLYBENCH_DUMP_FINISH;
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_ludcmp(int n,
		   DATA_TYPE POLYBENCH_2D(A,N,N,n,n),
		   DATA_TYPE POLYBENCH_1D(b,N,n),
		   DATA_TYPE POLYBENCH_1D(x,N,n),
		   DATA_TYPE POLYBENCH_1D(y,N,n))
{
  int i, j, k;

  DATA_TYPE w;

#pragma scop
    
    double *d_A,*d_B,*d_AT;
    double *d_b;
    const double cublas_alpha = 1.0;
    const double cublas_beta = 0.0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipMalloc((void **)&d_A, n * n * sizeof(double));
    hipMalloc((void **)&d_AT, n * n * sizeof(double));
    hipMalloc((void **)&d_b, n * sizeof(double));

    hipMemcpy(d_A, A, n * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(double), hipMemcpyHostToDevice);
    hipblasDgeam(handle,HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, &cublas_alpha, d_A, n, &cublas_beta, d_B, n, d_AT, n);

    int worksize = 0;
    int *devInfo;
    hipMalloc((void **)&devInfo, sizeof(int));
    hipsolverHandle_t solver_handle;
    hipsolverDnCreate(&solver_handle);

    hipsolverDnDgetrf_bufferSize(solver_handle, n, n, d_AT, n, &worksize);
    double *work;
    hipMalloc((void **)&work, worksize * sizeof(double));
    int *devIpiv;
    hipMalloc((void **)&devIpiv, n * sizeof(int));

    hipsolverDnDgetrf(solver_handle, n, n, d_AT, n, work, devIpiv, devInfo);
    int devInfo_h = 0;
    hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    if (devInfo_h != 0)
        printf("Unsuccessful getrf execution\n\n");
    printf("\nFactorized matrix\n");
    hipsolverDnDgetrs(solver_handle, HIPBLAS_OP_N, n, n, d_AT, n, devIpiv, d_b, n, devInfo);
    hipMemcpy(x, d_b, n * sizeof(double), hipMemcpyDeviceToHost);
   
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_AT);
    hipFree(d_b);
    hipFree(devInfo);
    hipFree(work);
    hipFree(devIpiv);
    hipsolverDnDestroy(solver_handle);

}


int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int n = N;

  /* Variable declaration/allocation. */
  POLYBENCH_2D_ARRAY_DECL(A, DATA_TYPE, N, N, n, n);
  POLYBENCH_1D_ARRAY_DECL(b, DATA_TYPE, N, n);
  POLYBENCH_1D_ARRAY_DECL(x, DATA_TYPE, N, n);
  POLYBENCH_1D_ARRAY_DECL(y, DATA_TYPE, N, n);


  /* Initialize array(s). */
  init_array (n,
	      POLYBENCH_ARRAY(A),
	      POLYBENCH_ARRAY(b),
	      POLYBENCH_ARRAY(x),
	      POLYBENCH_ARRAY(y));

  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  kernel_ludcmp (n,
		 POLYBENCH_ARRAY(A),
		 POLYBENCH_ARRAY(b),
		 POLYBENCH_ARRAY(x),
		 POLYBENCH_ARRAY(y));

  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  polybench_prevent_dce(print_array(n, POLYBENCH_ARRAY(x)));

  /* Be clean. */
  POLYBENCH_FREE_ARRAY(A);
  POLYBENCH_FREE_ARRAY(b);
  POLYBENCH_FREE_ARRAY(x);
  POLYBENCH_FREE_ARRAY(y);

  return 0;
}
