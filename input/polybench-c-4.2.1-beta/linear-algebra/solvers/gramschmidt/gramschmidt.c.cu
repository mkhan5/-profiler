
/* Header below added by Tulsi for replaced CUBLAS code */
#include "hip/hip_runtime.h"
#include <hipsolver.h>
#include <hipblas.h>



/**
 * This version is stamped on May 10, 2016
 *
 * Contact:
 *   Louis-Noel Pouchet <pouchet.ohio-state.edu>
 *   Tomofumi Yuki <tomofumi.yuki.fr>
 *
 * Web address: http://polybench.sourceforge.net
 */
/* gramschmidt.c: this file is part of PolyBench/C */

#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
#include <polybench.h>

/* Include benchmark-specific header. */
#include "gramschmidt.h"


/* Array initialization. */
static
void init_array(int m, int n,
		DATA_TYPE POLYBENCH_2D(A,M,N,m,n),
		DATA_TYPE POLYBENCH_2D(R,N,N,n,n),
		DATA_TYPE POLYBENCH_2D(Q,M,N,m,n))
{
  int i, j;

  for (i = 0; i < m; i++)
    for (j = 0; j < m; j++) {
      A[i][j] = (((DATA_TYPE) ((i*j) % m) / m )*100) + 10;
      Q[i][j] = 0.0;
    }
  for (i = 0; i < m; i++)
    for (j = 0; j < m; j++)
      R[i][j] = 0.0;
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int m, int n,
		 DATA_TYPE POLYBENCH_2D(A,M,N,m,n),
		 DATA_TYPE POLYBENCH_2D(R,N,N,n,n),
		 DATA_TYPE POLYBENCH_2D(Q,M,N,m,n))
{
  int i, j;

  POLYBENCH_DUMP_START;
  POLYBENCH_DUMP_BEGIN("R");
  for (i = 0; i < n; i++)
    for (j = 0; j < n; j++) {
	if ((i*n+j) % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
	fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, R[i][j]);
    }
  POLYBENCH_DUMP_END("R");

  POLYBENCH_DUMP_BEGIN("Q");
  for (i = 0; i < m; i++)
    for (j = 0; j < m; j++) {
	if ((i*n+j) % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
	fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, Q[i][j]);
    }
  POLYBENCH_DUMP_END("Q");
  POLYBENCH_DUMP_FINISH;
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
/* QR Decomposition with Modified Gram Schmidt:
 http://www.inf.ethz.ch/personal/gander/ */
static
void kernel_gramschmidt(int m, int n,
			DATA_TYPE POLYBENCH_2D(A,M,N,m,n),
			DATA_TYPE POLYBENCH_2D(R,N,N,n,n),
			DATA_TYPE POLYBENCH_2D(Q,M,N,m,n))
{
  int i, j, k;

  DATA_TYPE nrm;

//#pragma scop
    int i,j,k;
    double *d_A,*d_AT,*d_Q,*d_B;
    const double cublas_alpha = 1.0;
    const double cublas_beta = 0.0;
    hipblasHandle_t handle;

    hipblasCreate(&handle);
    hipMalloc((void **)&d_A,      m * m * sizeof(double));
    hipMalloc((void **)&d_Q,      m * m * sizeof(double));
    hipMemcpy(d_A, A, m * m * sizeof(double), hipMemcpyHostToDevice);
    hipMalloc((void **)&d_AT,      m * m * sizeof(double));

    hipblasDgeam(handle,HIPBLAS_OP_T, HIPBLAS_OP_N, m, m, &cublas_alpha, d_A, m, &cublas_beta, d_B, m, d_AT, m);

    int worksize = 0;
    int *devInfo;
    hipMalloc((void **)&devInfo, sizeof(int));
    hipsolverHandle_t solver_handle;
    hipsolverDnCreate(&solver_handle);

    hipsolverDnDgeqrf_bufferSize(solver_handle, m, m, d_AT, m, &worksize);
    double *work;
    hipMalloc((void **)&work, worksize * sizeof(double));
    double *TAU;
    hipMalloc((void **)&TAU, m * sizeof(double));

    hipsolverDnDgeqrf(solver_handle, m, m, d_AT, m, TAU, work, worksize, devInfo);

    int devInfo_h = 0;
    hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    if (devInfo_h != 0)
        printf("Unsuccessful geqrf execution\n\n");
    double *q_h;
    q_h = (double *)malloc( m*m*sizeof( double ));
    printf("\nFactorized matrix\n");

    hipblasDgeam(handle,HIPBLAS_OP_T, HIPBLAS_OP_N, m, m, &cublas_alpha, d_AT, m, &cublas_beta, d_B, m, d_A, m);
    hipMemcpy(R, d_A, m * m * sizeof(double), hipMemcpyDeviceToHost);
    for (i = 0; i < m; i++)
    {
        for (j = 0; j < m; j++)
        {
            q_h[i*m+j] = 0.0;
            if (i==j)
                q_h[i*m+j] = 1.0;
        }
    }

    hipMemcpy(d_Q, q_h, m * m * sizeof(double), hipMemcpyHostToDevice);
    hipsolverDnDormqr(solver_handle,HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, m, m, m, d_AT, m, TAU, d_Q, m, work, worksize, devInfo);

    devInfo_h = 0;
    hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    if (devInfo_h != 0)
        printf("Unsuccessful ormqr execution\n\n");
    hipMemcpy(Q, d_Q, m * m * sizeof(double), hipMemcpyDeviceToHost);
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_AT);
    hipFree(d_Q);
    hipFree(TAU);
    hipFree(work);
    hipFree(devInfo);
    hipsolverDnDestroy(solver_handle);

}


int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int m = M;
  int n = N;

  /* Variable declaration/allocation. */
  POLYBENCH_2D_ARRAY_DECL(A,DATA_TYPE,M,N,m,n);
  POLYBENCH_2D_ARRAY_DECL(R,DATA_TYPE,N,N,n,n);
  POLYBENCH_2D_ARRAY_DECL(Q,DATA_TYPE,M,N,m,n);

  /* Initialize array(s). */
  init_array (m, n,
	      POLYBENCH_ARRAY(A),
	      POLYBENCH_ARRAY(R),
	      POLYBENCH_ARRAY(Q));

  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  kernel_gramschmidt (m, n,
		      POLYBENCH_ARRAY(A),
		      POLYBENCH_ARRAY(R),
		      POLYBENCH_ARRAY(Q));

  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  polybench_prevent_dce(print_array(m, n, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(R), POLYBENCH_ARRAY(Q)));

  /* Be clean. */
  POLYBENCH_FREE_ARRAY(A);
  POLYBENCH_FREE_ARRAY(R);
  POLYBENCH_FREE_ARRAY(Q);

  return 0;
}
