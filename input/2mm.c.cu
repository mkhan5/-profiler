
/* Header below added by Tulsi for replaced CUBLAS code */
#include <hip/hip_runtime.h>
#include <hipblas.h>


#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
//#define N 3
int N=3;

void mm2( double A[N][N],
            double B[N][N],
            double C[N][N],
            double D[N][N],
            double tmp[N][N],
            double alpha,
            double beta
           )
{
  int i, j,k;

  /* D := alpha*A*B*C + beta*D */
    
    hipblasStatus_t status;
    hipblasHandle_t handle;
    double *d_A = 0;
    double *d_B = 0;
    double *d_C = 0;
    double *d_D = 0;
    double *d_tmpAB = 0;
    const double cublas_alpha = 1.0f;
    const double cublas_beta = 0.0f;
    hipblasCreate(&handle);
    
    hipMalloc((void **)&d_A, N * N * sizeof(d_A[0]));
    hipMalloc((void **)&d_B, N * N * sizeof(d_B[0]));
    hipMalloc((void **)&d_C, N * N * sizeof(d_C[0]));
    hipMalloc((void **)&d_D, N * N * sizeof(d_D[0]));
    hipMalloc((void **)&d_tmpAB, N * N * sizeof(d_tmpAB[0]));
    
    hipMemcpy(d_A, A,  N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N *N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, N *N * sizeof(double), hipMemcpyHostToDevice);
    
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &cublas_alpha, d_B, N, d_A, N, &cublas_beta, d_tmpAB, N);
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &cublas_alpha, d_C, N, d_tmpAB, N, &cublas_beta, d_D, N);
    hipMemcpy(D, d_D, N*N*sizeof(double), hipMemcpyDeviceToHost);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_D);
    hipFree(d_tmpAB);
    hipblasDestroy(handle);

}


int main(int argc, char** argv)
{

  int M = 3;


  double a[3][3] = { 1, 2, 5, 9, 23, -7, 6, 1, -3};
  double b[3][3] = { 17, 2, -1, -12, -19, -22, -1, 3, 9};
  double c[3][3] = {1, 5, -1, -2, 1, 21, -30, -11, -15};
  double tmp[3][3];
  double d[3][3];
  //sol: y = (13, 31, 49)
  double alpha = 1;
  double beta = 0;

  mm2(a, b, c, d, tmp, alpha, beta);
  //std::copy( res, res + 3, std::ostream_iterator<float>( std::cout, ","));
  int i,j;
  printf("The res x is \n");
  for( i = 0; i < N; i++)
    {
        for (j = 0; j < N; j++)
      {
        printf(" %f ",d[i][j]);
       }
       printf("\n");
    }

  printf("\n");

}
